/* FIXME
 * Correct, and refactor 'loop' to be a CUDA Kernel.
 * The new kernel should only do the work
 * of 1 iteration of the original loop.
 */


#include <hip/hip_runtime.h>
#include <cstdio>

void loop(int N)
{
  for (int i = 0; i < N; ++i)
  {
    printf("This is iteration number %d\n", i);
  }
}

int main()
{
  /*
   * When refactoring 'loop' to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * Use 1 block of threads.
   */

  int N = 10;
  loop(N);
}

